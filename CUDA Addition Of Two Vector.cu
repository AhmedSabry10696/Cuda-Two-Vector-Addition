#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <time.h>
using namespace std;

// function generate random numbers and assign it to array
void random_ints(int *a, int N)
{
	for (int i = 0; i < N; i++)
		a[i] = rand();
}

// create kernal "two vector addition"
__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

#define N 8                      // array size 
#define THREADS_PER_BLOCK 2

int main(void) {

	int *a, *b, *c;              // host data 
	int *d_a, *d_b, *d_c;		 // device data
	int size = N * sizeof(int);

	cout<<"\t\t\t*** CUDA TASK ***\n\t\t\t-----------------\n\n";

	// alloacate host data 
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	// allocate device data
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// copy data from host to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// call add kernal 
	add << < N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> > (d_a, d_b, d_c);

	// copy data back from device to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	cout<<"A\tB\tC\n------------------------\n";

	for (int i = 0; i < N; i++) {
		cout<<a[i]<<"\t"<<b[i]<<"\t"<<c[i]<<"\n";
		cout<<"------------------------\n";
	}

	// free allocated host data
	free(a);  free(b);  free(c);

	// free allocated device data 
	hipFree(d_a);  hipFree(d_b);  hipFree(d_c);

	return 0;
}
